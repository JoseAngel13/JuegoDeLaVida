#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <windows.h>
#define N 10000           //Tamaño del tablero potencias de 2 a partir de 16 
#define BLOCK_SIZE 16	
#define ITERACIONES 300	//numero de cambios totales del autómata


/*
 * Funcion para actualizar la matriz, intercambiandola entre la auxiliar
 * y la matriz principal.  
 */
__global__ void actualiza(int *malla, int *aux){
	int contador=0;
	int celActual;
	int i = blockDim.x * blockIdx.x +  threadIdx.x; //fila
    int j = blockDim.y * blockIdx.y + threadIdx.y; //Columna
	if (i < N && j < N) {
	celActual = i*N+j;
	//printf("%d ",celActual);
	//Izquierda Arriba
	if(i>0 && j>0 && malla[celActual-N-1]==1){
		contador++;
	}
	//Arriba
	if(i>0 && malla[celActual-N]==1){
		contador++;
	}
	//Arriba derecha
	if(i>0 && j<N-1 && malla[celActual+1-N]==1){
		contador++;
	}
	//Izquierda
	if(j>0 && malla[celActual-1]==1){
		contador++;
	}
	//Derecha
	if(j<N-1 && malla[celActual+1]==1){
		contador++;
	}
	//Abajo izquierda
	if(i<N-1 && j>0 && malla[celActual+N-1]==1){
		contador++;
	}
	//Abajo
	if(i<N-1 && malla[celActual+N]==1){
		contador++;
	}
	//Abajo derecha
	if(i<N-1 && j<N-1 && malla[celActual+1+N]==1){
		contador++;
	}

	if(malla[celActual]==1){		//Actuamos sobre las celulas en la copia de la matriz
		if(contador==2 || contador==3){//La celulas vivas con 2 o 3 celulas vivas pegadas, se mantiene vivas.
			aux[celActual]=1;
		}
		else{					//Si no se cumple la condicion, mueren.
			aux[celActual]=0;
			}
		}
	else{
		if(contador==3){		//Las celulas muertas con 3 celulas vivas pegadas, resucitan.
			aux[celActual]=1;
			}
		}
	contador=0;
	}


}

__global__ void copiaMatriz(int *malla, int *aux){
	int celActual;
	int i = blockDim.x * blockIdx.x +  threadIdx.x; //fila
    int j = blockDim.y * blockIdx.y + threadIdx.y; //Columna
	if (i < N && j < N) {
		celActual = i*N+j;
		aux[celActual] = malla[celActual];
	}
}


/*
 * Funcion para imprimir la matriz 
 * Como entradas son la malla y el tamaño
 */
__global__ void imprimeM(int *m){
//void imprimeM(int *m){
	for (int i = 0; i < N; ++i)
	{
		for (int j = 0; j < N; ++j)
		{
			if (m[i*N+j]==1)
			{
				printf("* ");
			}
			else{
				printf("- ");
			}
		}
		printf("\n");
	}
}


int main() {
    //Cambiar tipos de datos
    //int float double long
    int *tablero, *tablero_aux;
	int *d_tablero, *d_tablero_aux;
	clock_t  inicio, final;
    size_t size = N*N * sizeof(int);

    //Asignacion de memoria del lado del host
    tablero = (int*)malloc(size);
    tablero_aux = (int*)malloc(size);
    

    //Asignacion de memoria del lado de device
    hipMalloc(&d_tablero, size);
    hipMalloc(&d_tablero_aux, size);
	
	FILE * archivo = fopen("../mat5000.txt", "r");
	if (archivo==NULL) {fputs ("File error",stderr); exit (1);}

	char caracterAuxiliar;

	for(int i=0;i<N;i++){
		for(int j=0;j<N;j++){
			caracterAuxiliar = fgetc(archivo);
			if (caracterAuxiliar == '1'){
				tablero_aux[i*N+j]=tablero[i*N+j]=1;
			}
			else {
				tablero_aux[i*N+j]=tablero[i*N+j]=0;
			}
		}
	}

	fclose(archivo);
    
	printf("\n");


    hipMemcpy(d_tablero,tablero,size,hipMemcpyHostToDevice);
    hipMemcpy(d_tablero_aux,tablero_aux,size,hipMemcpyHostToDevice);


    //VOLVER CONSTANTE
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((N + dimBlock.x-1)/dimBlock.x, (N+dimBlock.y-1)/dimBlock.y);
    
    /*
    * Repite el proceso el mismo número de veces.
    *  
    */
	//tiempo inicial
	inicio = clock();
	//imprimeM<<<1,1>>>(d_tablero);
	hipDeviceSynchronize();
	for (int i = 0; i < ITERACIONES; ++i)
	{
	 	//printf("Iteracion %d\n",i+1);
		actualiza<<<dimGrid,dimBlock>>>(d_tablero,d_tablero_aux);
		hipDeviceSynchronize();//espera a que todos los hilos terminen su ejecución
		copiaMatriz<<<dimGrid,dimBlock>>>(d_tablero_aux,d_tablero);
		hipDeviceSynchronize();
		//imprimeM<<<1,1>>>(d_tablero);
		//hipDeviceSynchronize();
	}
	final = clock();
	double tiempo = ((double)final - inicio) / CLOCKS_PER_SEC;
	printf("el tiempo final es %f", tiempo);
	//timepo final

    free(tablero);
    free(tablero_aux);

    hipFree(d_tablero);
    hipFree(d_tablero_aux);

    return 0;
}