#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include<time.h>
#include <windows.h>
#define N 32           //Tamaño del tablero potencias de 2 a partir de 16 
#define BLOCK_SIZE 16	
#define ITERACIONES 100	//numero de cambios totales del autómata


/*
 * Funcion para actualizar la matriz, intercambiandola entre la auxiliar
 * y la matriz principal.  
 */
__global__ void actualiza(int *malla, int *aux){
	int contador=0;
	int celActual;
	int i = blockDim.x * blockIdx.x +  threadIdx.x; //fila
    int j = blockDim.y * blockIdx.y + threadIdx.y; //Columna
	if (i < N && j < N) {
	celActual = i*N+j;

	//Izquierda Arriba
	if(i>0 && j>0 && malla[celActual-N-1]==1){
		contador++;
	}
	//Arriba
	if(i>0 && malla[celActual-N]==1){
		contador++;
	}
	//Arriba derecha
	if(i>0 && j<N-1 && malla[celActual+1-N]==1){
		contador++;
	}
	//Izquierda
	if(j>0 && malla[celActual-1]==1){
		contador++;
	}
	//Derecha
	if(j<N-1 && malla[celActual+1]==1){
		contador++;
	}
	//Abajo izquierda
	if(i<N-1 && j>0 && malla[celActual+N-1]==1){
		contador++;
	}
	//Abajo
	if(i<N-1 && malla[celActual+N]==1){
		contador++;
	}
	//Abajo derecha
	if(i<N-1 && j<N-1 && malla[celActual+1+N]==1){
		contador++;
	}

	if(malla[celActual]==1){		//Actuamos sobre las celulas en la copia de la matriz
		if(contador==2 || contador==3){//La celulas vivas con 2 o 3 celulas vivas pegadas, se mantiene vivas.
			aux[celActual]=1;
		}
		else{					//Si no se cumple la condicion, mueren.
			aux[celActual]=0;
			}
		}
	else{
		if(contador==3){		//Las celulas muertas con 3 celulas vivas pegadas, resucitan.
			aux[celActual]=1;
			}
		}
	contador=0;
	
	malla[celActual] = aux[celActual];//Copiamos la matriz origen en destino
	}
}


/*
 * Funcion para imprimir la matriz 
 * Como entradas son la malla y el tamaño
 */
__global__ void imprimeM(int *m){
	for (int i = 0; i < N; ++i)
	{
		for (int j = 0; j < N; ++j)
		{
			if (m[i*N+j]==1)
			{
				printf("* ");
			}
			else{
				printf("- ");
			}
		}
		printf("\n");
	}
}

/*  buscar esta funcion para llenar las matricesde forma aleatoria
    arc4random(); 
    arc4random solo se puede usar en C con un compilador anterior a c99
*/


int main() {
    //Cambiar tipos de datos
    //int float double long
    int *tablero, *tablero_aux;
    int *d_tablero, *d_tablero_aux;
    size_t size = N*N * sizeof(int);

    //Asignacion de memoria del lado del host
    tablero = (int*)malloc(size);
    tablero_aux = (int*)malloc(size);
    

    //Asignacion de memoria del lado de device
    hipMalloc(&d_tablero, size);
    hipMalloc(&d_tablero_aux, size);

    //Llenado de las matrices en forma secuencial
    //srand ((int)time(NULL));
	
	/*for(int i=0; i<N*N;i++)
    {   
		tablero_aux[i] = tablero[i] = (float)(rand()%2);
        //printf("%ld %ld\n",tablero[i],tablero_aux[i]);
	}*/
	
	FILE * archivo = fopen("../mat1000.txt", "r");
	if (archivo==NULL) {fputs ("File error",stderr); exit (1);}

	char caracterAuxiliar;

	for(int i=0;i<N;i++){
		for(int j=0;j<N;j++){
			caracterAuxiliar = fgetc(archivo);
			if (caracterAuxiliar == '1'){
				tablero_aux[i*N+j]=tablero[i*N+j]=1;
			}
			else {
				tablero_aux[i*N+j]=tablero[i*N+j]=0;
			}
		}
	}

	fclose(archivo);
    
	//imprimeM(tablero);
	printf("\n");
	//imprimeM(tablero_aux);


    hipMemcpy(d_tablero,tablero,size,hipMemcpyHostToDevice);
    hipMemcpy(d_tablero_aux,tablero_aux,size,hipMemcpyHostToDevice);


    //VOLVER CONSTANTE
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((N + dimBlock.x-1)/dimBlock.x, (N+dimBlock.y-1)/dimBlock.y);

    //actualiza<<<dimGrid,dimBlock>>>(d_tablero,d_tablero_aux);
    //hipDeviceSynchronize();//espera a que todos los hilos terminen su ejecución
    
    /*
    * Repite el proceso el mismo número de veces.
    *  
    */
	//tiempo inicial
	for (int i = 0; i < ITERACIONES; ++i)
	{
	 	printf("Iteracion %d\n",i+1);

		actualiza<<<dimGrid,dimBlock>>>(d_tablero,d_tablero_aux);
		hipDeviceSynchronize();//espera a que todos los hilos terminen su ejecución
		//hipMemcpy(tablero,d_tablero,size,hipMemcpyDeviceToHost);
		imprimeM<<<1,1>>>(d_tablero);
	}
    //hipMemcpy(resultado,d_resultado,size,hipMemcpyDeviceToHost);
	//timepo final

    free(tablero);
    free(tablero_aux);

    hipFree(d_tablero);
    hipFree(d_tablero_aux);

    return 0;

}